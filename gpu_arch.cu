
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int main(void) {

    int dev=0;
    char str[100];

    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    // print architecture
    sprintf(str,"export GPU_ARCH=%d%d",deviceProp.major, deviceProp.minor);
    printf("%s\n",str);
}
