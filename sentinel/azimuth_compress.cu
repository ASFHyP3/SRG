#include "hip/hip_runtime.h"
//!!!!!!!!!!!!!!!
//!
//!
//!  azimuth compression subroutine for use in Sentinel back projection processor
//!    called from a fortran main program
//!
//!
//!!!!!!!!!!!!!!

#include <stdlib.h>
#include <stdio.h>
#include <complex.h>
#include <math.h>
#include <unistd.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <omp.h>
#include <fcntl.h>
#include <time.h>
#include <sys/time.h>

struct ellipsoid{
  double a;
  double e2;
};

extern "C" void latlon_ ( ellipsoid *elp, double *xyz, double *llh,int *i_type );
extern "C" void orbitrangetime_(double *xyz,double *timeorbit,double *xx, double *vv,int *numstatevec,double *tmid,double *satx, double *satv,double *tline,double *rngpix);

// pixelint - the routine to compute complex sum in gpu

__global__ void pixelint(hipFloatComplex *burstdata_d, double *satloc_d, double *xyz_d, double *azoff_d, int demwidth_d, int aperture_d, int iaperture_d, int rawdatalines_d, int samplesPerBurst_d, double rngstart_d, double rngend_d,  double dmrg_d, double wvl_d, hipFloatComplex *outdata_d, int nlines_d){

  // set up gpu grid parameters
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  //printf("index stride %d %d\n",index,stride);

  // internal variables for integration
  hipDoubleComplex cacc, cphase;
  hipFloatComplex cval;
  int intr;
  double range, fracr,phase,r,vectx,vecty,vectz,pi;
  int azline;
  int azstart, azend;

  pi=4.*atan2(1.,1.);

  // loop over 1D version of 2D arrays
  for (long loop = index; loop < nlines_d * demwidth_d; loop +=stride){

    cacc = make_hipDoubleComplex(0.,0.);
    if (azoff_d[loop] > 0.){
      azstart=round(azoff_d[loop] - aperture_d / 2);
      azend=round(azoff_d[loop] + aperture_d / 2);
      azstart=max(azstart,0);
      azend=min(azend,rawdatalines_d-1);
      for (azline=azstart; azline<azend; azline++){
	vectx=xyz_d[loop * 3 + 0]-satloc_d[3*azline];
	vecty=xyz_d[loop * 3 + 1]-satloc_d[3*azline+1];
	vectz=xyz_d[loop * 3 + 2]-satloc_d[3*azline+2];
	range=sqrt(vectx*vectx+vecty*vecty+vectz*vectz);
	if (range>=rngstart_d && range <= rngend_d - dmrg_d){
	  r=(range - rngstart_d) / dmrg_d ;
	  intr=floor(r);
	  fracr=r-intr;

	  cval=hipCaddf( hipCmulf(burstdata_d[azline * samplesPerBurst_d + intr], make_hipComplex(1-fracr,0.)), hipCmulf( burstdata_d[azline * samplesPerBurst_d + (intr+1)], make_hipFloatComplex(fracr,0.)));

	  phase = 4. * pi / wvl_d * range;
	  cphase = make_hipDoubleComplex(cos(phase),sin(phase));
	  cacc = hipCadd(cacc, hipCmul(hipComplexFloatToDouble(cval), cphase));
	}  // end range test
      }  // end azline loop
    } // end pixel test

    outdata_d[loop]=hipComplexDoubleToFloat(cacc);
    outdata_d[loop]=hipCmulf(outdata_d[loop],make_hipFloatComplex(1./(azend-azstart),0.)); // part of sigma calibration for az integratiom length
  } // end loop loop

}

//  set xyz array
__global__ void setxyz(short *demin_d, double *xyz_d, double *xyzfit_d, double *azoff_d, int demwidth_d, double firstlat_d, double deltalat_d, double firstlon_d, double deltalon_d, int firstline_d, int nlines_d, int firstpix_d, int lastpix_d){

  // local array definitions for loop
  double *llh = (double *) malloc(sizeof(double)*3);
  double *xyztemp = (double *) malloc(sizeof(double)*3);
  double *satx = (double *) malloc(sizeof(double)*3);
  double *satv = (double *) malloc(sizeof(double)*3);
  double *unitlookvector = (double *) malloc(sizeof(double)*3);
  //double * lon;
  //hipMalloc (&lon, sizeof(double) * demwidth_d);

  //  struct ellipsoid elp = { 6378137.0, 0.0066943799901499996};

  // set up gpu grid parameters
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  double pi = 4.0*atan2(1.,1.);
  double deg2rad = pi / 180.;
  double a = 6378137.0;
  double e2 = 0.0066943799901499996;
  double re;
  int line, pixel;
  double lat;
  long loop;
  long long xyzoffset;
  double llhlat, llhlon, llhhgt;

  // loop over full array
  for (loop = index; loop < demwidth_d * nlines_d; loop+=stride){
    line = int (loop / demwidth_d);
    pixel = loop - line * demwidth_d;
    lat = firstlat_d + (line + firstline_d) * deltalat_d;

    if (pixel >= firstpix_d && pixel <= lastpix_d){

      xyzoffset = ((long long int) line * (long long int) demwidth_d + (long long int) pixel ) * (long long int) 3;

      llhlat = lat * deg2rad;
      llhlon = (firstlon_d + pixel * deltalon_d) * deg2rad;
      llhhgt = demin_d[xyzoffset / 3];

      re = a/sqrt(1.0 - e2*sin(llhlat)*sin(llhlat));
          
      xyz_d[xyzoffset+0] = (re + llhhgt)*cos(llhlat)*cos(llhlon);
      xyz_d[xyzoffset+1] = (re + llhhgt)*cos(llhlat)*sin(llhlon);
      xyz_d[xyzoffset+2] = (re - (re * e2) + llhhgt)*sin(llhlat);        

      if(pixel == firstpix_d){
        xyzfit_d[line * 9 + 0]=xyz_d[xyzoffset+0];
        xyzfit_d[line * 9 + 1]=xyz_d[xyzoffset+1];
        xyzfit_d[line * 9 + 2]=xyz_d[xyzoffset+2];
        }
      if(pixel == firstpix_d + int((lastpix_d-firstpix_d)/2-1)){
        xyzfit_d[line * 9 + 3]=xyz_d[xyzoffset+0];
        xyzfit_d[line * 9 + 4]=xyz_d[xyzoffset+1];
        xyzfit_d[line * 9 + 5]=xyz_d[xyzoffset+2];
        }
      if(pixel == firstpix_d + 2 * int((lastpix_d-firstpix_d)/2-1)){
        xyzfit_d[line * 9 + 6]=xyz_d[xyzoffset+0];
        xyzfit_d[line * 9 + 7]=xyz_d[xyzoffset+1];
        xyzfit_d[line * 9 + 8]=xyz_d[xyzoffset+2];
	}

    }  // end pixel test
  }  // end loop loop
}  // end routine setxyz

//  set azoff array
__global__ void setazoff(double *coef_d, double *azoff_d, int demwidth_d, int nlines_d, int firstpix_d, int lastpix_d, int aperture_d, int rawdatalines_d){

  // set up gpu grid parameters
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  int line, pixel;
  double fit, arg;
  long loop;

  for (loop = index; loop < demwidth_d * nlines_d; loop+=stride){
    line = int (loop / demwidth_d);
    pixel = loop - line * demwidth_d;
//    printf("line pixel %d %d\n",line, pixel);

    if (pixel >= firstpix_d && pixel <= lastpix_d){
      arg=(float(pixel-firstpix_d)/float(lastpix_d-firstpix_d)*2.-1.);
      fit=coef_d[line*3+0]*arg*arg+coef_d[line*3+1]*arg+coef_d[line*3+2];
      azoff_d[line * demwidth_d + pixel]=-1;
      if (fit > aperture_d/2 && fit < rawdatalines_d- aperture_d/2){
        azoff_d[line * demwidth_d + pixel]=fit;
      }  // end fit in aperture test
  }  // end pixel test
  }  // end loop loop
}  //end setazoff routine


extern "C" void azimuth_compress_(
				  _Complex float *burstdata,
				  double *satloc,
				  int *rawdatalines,
				  int *samplesPerBurst,
				  int *demwidth,
				  int *demlength,
				  int *fdout,
				  int *fddem,
				  double *deltalat,
				  double *deltalon,
				  double *firstlat,
				  double *firstlon,
				  double *latlons,
				  double *timeorbit,
				  double *xx,
				  double *vv,
				  int *numstatevec,
				  double *rngstart,
				  double *rngend,
				  double *tstart,
				  double *tend,
				  double *tmid,
				  double *xyz_mid,
				  double *vel_mid,
				  double *t,
				  double *dtaz,
				  double *dmrg,
				  double *wvl,
				  int *aperture,
				  int *iaperture,
				  double *angc0,
				  double *angc1,
				  double *prf)
{

  // internal variables
  double *lon;
  short *demin;
//  double *azoff;
  double tline;
  double rngpix;
  //double *xyz;
  double *xyzfit;
  double *coef;
  double umag, fd, veff, td, udotv;
  int firstline, lastline;  // limits on line loop
  int firstpix, lastpix;  // limits on pixel loop
  long long int arraysize;
  FILE *fpout; // stream for file descriptor *fdout

  int naperture; // naperture is integration midpoint in pixels
  int y1,y2,y3;
  int pixel,line,i;
  long long int nbytes;
  int nlines;
  off_t iaddr_off_t;
  size_t iaddr_size_t;
  _Complex float *outdata, *indata;

  // variables for openmp pragma argument passing
//  int ompdemwidth=*demwidth;
  double omptmid=*tmid;
  double ompangc0=*angc0, ompangc1=*angc1;
  double ompprf=*prf;
  int omprawdatalines=*rawdatalines;
//  int ompaperture=*aperture;
  double ompwvl=*wvl;

//  struct timeval t0;
//  double time0,time1;

//  gettimeofday(&t0, NULL);
//  time0=t0.tv_sec+t0.tv_usec/1.e6;

  // set a gpudevice

  long int getgpu = hipSetDevice(0);
//  printf("GPU set return: %d\n",getgpu);
  if (getgpu != 0){
     printf("Can't grab GPU %ld\n",getgpu);
     FILE *fgetgpu = NULL;
     fgetgpu =fopen("getgpulog","a");
     fprintf(fgetgpu,"Can't grab GPU %ld\n",getgpu);
     fclose(fgetgpu);
     }    


/*
// list some gpu attributes to see where we connected
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);	
    printf("  Device name: %s\n", prop.name);
}
*/

//  int *gpudevice;
//  long int getdevice = hipGetDevice(*gpudevice);
//  printf("Using gpu %d\n",gpudevice);
 
//  gettimeofday(&t0, NULL);
//  time1=t0.tv_sec+t0.tv_usec/1.e6;
//  printf(" grab a gpu time %9.3f\n",time1-time0);

//  gettimeofday(&t0, NULL);
//  time0=t0.tv_sec+t0.tv_usec/1.e6;

  // how much memory will we need to allocate?
//  printf("latlons %f %f %f %f\n",latlons[0],latlons[1],latlons[2],latlons[3]);
  lastline=int((latlons[0] - *firstlat) / *deltalat );
  firstline=int((latlons[1] - *firstlat) / *deltalat );
  if (firstline < 0) firstline=0;
  if (lastline < 0) lastline=0;
  if (firstline > *demlength-1) firstline= *demlength -1;
  if (lastline > *demlength-1) lastline= *demlength -1;
  nlines=lastline-firstline+1;
  printf("Burst line limits, size (first,last,nlines) %d %d %d\n",firstline,lastline,nlines);

  // malloc cpu arrays
  arraysize = (long int) nlines * (long int) *demwidth;
  // printf("arraysize nlines demwidth: %ld %d %d\n",arraysize, nlines, *demwidth);
lon = (double *) malloc(*demwidth * sizeof(double));
  demin = (short *) malloc(arraysize * sizeof(short));
//  azoff = (double *) malloc(arraysize * sizeof(double));
  //  pixeltime = (double *) malloc(*demwidth * sizeof(double));
  // xyz = (double *) malloc(arraysize * sizeof(double) * 3);
  xyzfit = (double *) malloc(nlines * sizeof(double) * 9);
  coef = (double *) malloc(nlines * sizeof(double) * 3);
  outdata = (_Complex float *)malloc(arraysize * sizeof(_Complex float));
  indata = (_Complex float *)malloc(arraysize * sizeof(_Complex float));
  // printf("bytes for indata outdata %lld\n",arraysize*8);
  
//  gettimeofday(&t0, NULL);
//  time1=t0.tv_sec+t0.tv_usec/1.e6;
//  printf(" cpu malloc time %9.3f\n",time1-time0);

//  gettimeofday(&t0, NULL);
//  time0=t0.tv_sec+t0.tv_usec/1.e6;

  // gpu array definitions
  hipFloatComplex *burstdata_d;
  double *satloc_d;
  double *azoff_d, *xyz_d;
  double *xyzfit_d;
  double *coef_d;
  hipFloatComplex *outdata_d;
  short *demin_d;

  hipMalloc( (void **)&burstdata_d, sizeof(_Complex float) * *rawdatalines * *samplesPerBurst);
  hipMalloc( (void **)&satloc_d, sizeof(double) * 3 * *rawdatalines);
  hipMalloc( (void **)&azoff_d, sizeof(double) * arraysize);
  hipMalloc( (void **)&xyz_d, sizeof(double) * 3 * arraysize);
  hipMalloc( (void **)&xyzfit_d, sizeof(double) * 9 * nlines);
  hipMalloc( (void **)&coef_d, sizeof(double) * 3 * nlines);
  hipMalloc( (void **)&outdata_d, sizeof(_Complex float) * arraysize);
  hipMalloc( (void **)&demin_d, sizeof(short) * arraysize);

//  gettimeofday(&t0, NULL);
//  time1=t0.tv_sec+t0.tv_usec/1.e6;
//  printf(" gpu malloc time %9.3f\n",time1-time0);

//  gettimeofday(&t0, NULL);
//  time0=t0.tv_sec+t0.tv_usec/1.e6;

  // constants and such
  double pi;
  pi = 4. * atan2(1.0,1.0);

  //  get some basic gpu architecture info
  int blockSize = 256;
  int numBlocks = (*demwidth+blockSize-1)/blockSize;
//  printf("GPU blocksize, numblocks: %d %d\n",blockSize,numBlocks);

//  gettimeofday(&t0, NULL);
//  time1=t0.tv_sec+t0.tv_usec/1.e6;
//  printf(" set constants time %9.3f\n",time1-time0);

//  gettimeofday(&t0, NULL);
//  time0=t0.tv_sec+t0.tv_usec/1.e6;

  //  start setting up shared data arrays in gpu, transfer raw data and satellite locations
  hipMemcpy( burstdata_d, burstdata, sizeof(_Complex float) * *rawdatalines * *samplesPerBurst, hipMemcpyHostToDevice );
  hipMemcpy( satloc_d, satloc, sizeof(double) * *rawdatalines * 3, hipMemcpyHostToDevice );

//  gettimeofday(&t0, NULL);
//  time1=t0.tv_sec+t0.tv_usec/1.e6;
//  printf(" transfer burst and satloc to gpu time %9.3f\n",time1-time0);

//  gettimeofday(&t0, NULL);
//  time0=t0.tv_sec+t0.tv_usec/1.e6;

  // set up longitude loop
  firstpix=(latlons[2]-*firstlon)/ *deltalon; if (firstpix < 0)firstpix=0;
  lastpix=(latlons[3]-*firstlon)/ *deltalon; if (lastpix > *demwidth)lastpix= *demwidth;

  //  define longitude array
  for (i=0;i<*demwidth;i++){
    lon[i]=*firstlon + (i-1)* *deltalon;
  }

  // zero out data array before integration
  for (int j=0; j<nlines; j++){
    for (i=0;i<*demwidth;i++){
      outdata[i+j * *demwidth]=0.+0.*i;
    }}

//  gettimeofday(&t0, NULL);
//  time1=t0.tv_sec+t0.tv_usec/1.e6;
//  printf(" initialize lon and outdata array time %9.3f\n",time1-time0);

//  gettimeofday(&t0, NULL);
//  time0=t0.tv_sec+t0.tv_usec/1.e6;

  //  process full burst, begin by grabbing proper section of DEM
  iaddr_off_t=(long long int) firstline * (long long int) *demwidth * (long long int) 2;
  nbytes=lseek(*fddem, iaddr_off_t, SEEK_SET);
  iaddr_size_t= (long long int) nlines * (long long int) *demwidth * (long long int) 2;
  nbytes=read(*fddem,demin,iaddr_size_t);
  if (nbytes < 0) {
    printf("dem read error %lld\n",nbytes);
    printf("iaddr_off_t %ld, iaddr_size_t %ld\n",iaddr_off_t, iaddr_size_t);
    printf("nlines %d, demwidth %d\n",nlines,*demwidth);
  }
         
//  gettimeofday(&t0, NULL);
//  time1=t0.tv_sec+t0.tv_usec/1.e6;
//  printf(" DEM section read in time %9.3f\n",time1-time0);

//  gettimeofday(&t0, NULL);
//  time0=t0.tv_sec+t0.tv_usec/1.e6;

  //  set xyz array in gpu
  hipMemcpy( demin_d, demin, sizeof(short) * arraysize, hipMemcpyHostToDevice );
  setxyz<<< numBlocks,blockSize >>>(demin_d, xyz_d, xyzfit_d, azoff_d, *demwidth, *firstlat, *deltalat, *firstlon, *deltalon, firstline, nlines, firstpix, lastpix);
  hipDeviceSynchronize();
  hipMemcpy( xyzfit, xyzfit_d, sizeof(double) * nlines * 9, hipMemcpyDeviceToHost );

//  gettimeofday(&t0, NULL);
//  time1=t0.tv_sec+t0.tv_usec/1.e6;
//  printf(" compute xyz in gpu time %9.3f\n",time1-time0);

//  gettimeofday(&t0, NULL);
//  time0=t0.tv_sec+t0.tv_usec/1.e6;

  // remainder of azoff calcs before integration

  double *xyztemp = (double *) malloc(sizeof(double)*3);
  double *satx = (double *) malloc(sizeof(double)*3);
  double *satv = (double *) malloc(sizeof(double)*3);
  double *unitlookvector = (double *) malloc(sizeof(double)*3);

  //  loop over lines in DEM for burst
  for (line=firstline; line<lastline;line++){

    for (int ipix=0; ipix<3; ipix++){
      pixel = firstpix + ipix * int((lastpix - firstpix)/2 - 1);

      //xyzoffset = ((long long int) (line-firstline) * (long long int) ompdemwidth + (long long int) pixel ) * (long long int) 3;

      // for start, middle, and end get integration midpoints

      xyztemp[0] = xyzfit[(line-firstline)*9 + ipix * 3 + 0];
      xyztemp[1] = xyzfit[(line-firstline)*9 + ipix * 3 + 1];
      xyztemp[2] = xyzfit[(line-firstline)*9 + ipix * 3 + 2];

      tline = omptmid;
      satx[0] = xyz_mid[0];satx[1] = xyz_mid[1];satx[2] = xyz_mid[2];
      satv[0] = vel_mid[0];satv[1] = vel_mid[1];satv[2] = vel_mid[2];
      // get the zero doppler location of the satellite
      orbitrangetime_(xyztemp,timeorbit,xx,vv,numstatevec,&omptmid,satx,satv,&tline,&rngpix);

      // tops-specific geometry calculations
      unitlookvector[0]=(xyztemp[0]-satx[0]); unitlookvector[1]=(xyztemp[1]-satx[1]); unitlookvector[2]=(xyztemp[2]-satx[2]);
      umag=sqrt(unitlookvector[0]*unitlookvector[0]+unitlookvector[1]*unitlookvector[1]+unitlookvector[2]*unitlookvector[2]);
      unitlookvector[0]=unitlookvector[0]/umag; unitlookvector[1]=unitlookvector[1]/umag; unitlookvector[2]=unitlookvector[2]/umag;
      udotv=satv[0]*unitlookvector[0]+satv[1]*unitlookvector[1]+satv[2]*unitlookvector[2];
      fd=(2.0/ ompwvl) * udotv;
      veff=sqrt(satv[0]*satv[0]+satv[1]*satv[1]+satv[2]*satv[2]);
      td=(rngpix* (ompwvl)/2./veff/veff*fd-rngpix*(ompangc0)*pi/180./veff)/(1.+rngpix*(ompangc1)*pi/180./veff);
      naperture=td*(ompprf) + omprawdatalines/2;

      if (ipix == 0)y1=naperture;
      if (ipix == 1)y2=naperture;
      if (ipix == 2)y3=naperture;
      // if at last pixel, save coefficients for azoff fit
      if (ipix==2){
         coef[(line-firstline)*3+0]=0.5*(y1+y3-2.*y2);
         coef[(line-firstline)*3+1]=0.5*(y3-y1);
         coef[(line-firstline)*3+2]=y2;
         //a=0.5*(y1+y3-2.*y2);
         //b=0.5*(y3-y1);
         //c=y2;
         }

    } // end pixel loop
  }  // end line loop

// compute azoff array in gpu, parabolas for each line

   hipMemcpy( coef_d, coef, sizeof(double) * nlines * 3, hipMemcpyHostToDevice );
   setazoff<<< numBlocks,blockSize >>>(coef_d, azoff_d, *demwidth, nlines, firstpix, lastpix, *aperture, *rawdatalines);

      free(xyztemp);
      free(satx);
      free(satv);
      free(unitlookvector);

//  gettimeofday(&t0, NULL);
//  time1=t0.tv_sec+t0.tv_usec/1.e6;
//  printf(" time in cpu azoff loop %9.3f\n",time1-time0);

//  gettimeofday(&t0, NULL);
//  time0=t0.tv_sec+t0.tv_usec/1.e6;

    pixelint<<< numBlocks,blockSize >>>(burstdata_d,satloc_d,xyz_d,azoff_d, *demwidth, *aperture, *iaperture, *rawdatalines, *samplesPerBurst, *rngstart, *rngend, *dmrg, *wvl, outdata_d, nlines);

  hipDeviceSynchronize();
//  gettimeofday(&t0, NULL);
//  time1=t0.tv_sec+t0.tv_usec/1.e6;
//  printf(" time in gpu pixel integration  %9.3f\n",time1-time0);

//  gettimeofday(&t0, NULL);
//  time0=t0.tv_sec+t0.tv_usec/1.e6;

  hipMemcpy( outdata, outdata_d, sizeof(_Complex float) * arraysize, hipMemcpyDeviceToHost );

  // get existing data for that burst
  iaddr_off_t=(long long int) firstline * (long long int) *demwidth * (long long int) 8;
  iaddr_size_t= arraysize * 8;

  //printf("file descriptor: *fdout fdout %lld %lld\n",*fdout,fdout);
  // stream for *fdout file descriptor
  fpout = fdopen(*fdout,"r+");
  // nbytes=lseek(*fdout, iaddr_off_t, SEEK_SET);
  nbytes=fseek(fpout, iaddr_off_t, SEEK_SET);
  //printf("seek nbytes iaddr_off_t %lld %lld\n",nbytes,iaddr_off_t);
  // nbytes=read(*fdout,indata,iaddr_size_t);
  nbytes=fread(indata, 1, iaddr_size_t, fpout);
  //printf("iaddr_off_t iaddr_size_t nbytes arraysize %lld %lld %lld %lld\n",iaddr_off_t,iaddr_size_t,nbytes,arraysize);
//  printf(" pointer %lld\n",line * *demwidth + *demwidth);
  // update if pixel computed
  for (line=0; line < nlines; line++){
    for (pixel=0; pixel< *demwidth; pixel++){
      if(abs(creal(outdata[line * *demwidth +pixel])) > 1.e-18)indata[line * *demwidth + pixel]=outdata[line * *demwidth + pixel];
    }
  }
  // write line to file
  //nbytes=lseek(*fdout, iaddr_off_t, SEEK_SET);
  nbytes=fseek(fpout, iaddr_off_t, SEEK_SET);
  // printf("seek 2 nbytes %lld\n",nbytes);
  //nbytes=write(*fdout,indata,iaddr_size_t);
  nbytes=fwrite(indata, 1, iaddr_size_t, fpout);
  //  printf("iaddr_off_t iaddr_size_t nbytes arraysize %lld %lld %lld %lld\n",iaddr_off_t,iaddr_size_t,nbytes,arraysize);

//  gettimeofday(&t0, NULL);
//  time1=t0.tv_sec+t0.tv_usec/1.e6;
//  printf(" update and output write time %9.3f\n",time1-time0);

  // free up memory in gpu
  hipFree(burstdata_d);
  hipFree(satloc_d);
  hipFree(azoff_d);
  hipFree(xyz_d);
  hipFree(xyzfit_d);
  hipFree(coef_d);
  hipFree(outdata_d);
  hipFree(demin_d);

  // free up cpu memory
  free(indata);
  free(outdata);
  free(demin);
  free(xyzfit);
  free(coef);
  free(lon);
  
}

