//!!!!!!!!!!!!!!
//!
//!   getarch - get gpu/cpu architecture of machine
//!
//!!!!!!!!!!!!!!

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <omp.h>

extern "C" void getarch_(long *numgpus, long *numcpus)
{

// get the configuration of the machine
  int nDevices, iDevice;  // multigpu handling parameters
  hipGetDeviceCount(&nDevices);
  printf("Number of gpus: %d\n",nDevices);
  printf("Number of cpus: %d\n",omp_get_num_procs());
  for(iDevice=0;iDevice<nDevices;iDevice++){
    hipDeviceProp_t dprop;
    hipGetDeviceProperties(&dprop,iDevice);
    printf(" %d %s\n",iDevice,dprop.name);
  }
  *numgpus=nDevices;
  *numcpus=omp_get_num_procs();
}

