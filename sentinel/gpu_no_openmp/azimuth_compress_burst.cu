#include "hip/hip_runtime.h"
//!!!!!!!!!!!!!!
//!
//!
//!  azimuth compression subroutine for use in Sentinel back projection processor
//!    called from a fortran main program
//!
//!
//!!!!!!!!!!!!!!

#include <stdlib.h>
#include <stdio.h>
#include <complex.h>
#include <math.h>
#include <unistd.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <omp.h>

struct ellipsoid{
  double a;
  double e2;
};

extern "C" void latlon_ ( ellipsoid *elp, double *xyz, double *llh,int *i_type );
extern "C" void orbitrangetime_(double *xyz,double *timeorbit,double *xx, double *vv,int *numstatevec,double *tmid,double *satx, double *satv,double *tline,double *rngpix);

// pixelint - the routine to compute complex sum in gpu

__global__ void pixelint(hipFloatComplex *burstdata_d, double *satloc_d, double *xyz_d, double *azoff_d, int *demwidth_d, int *aperture_d, int *iaperture_d, int *rawdatalines_d, int *samplesPerBurst_d, double *rngstart_d, double *rngend_d,  double *pixeltime_d, double *dmrg_d, double *wvl_d, hipFloatComplex *outdata_d){

// set up gpu grid parameters
int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;
//printf("index stride %d %d\n",index,stride);

// internal variables for integration
hipDoubleComplex cacc, cphase;
hipFloatComplex cval;
int intr;
double range, fracr,phase,r,vectx,vecty,vectz,pi;
int azline;
int azstart, azend;

pi=4.*atan2(1.,1.);

/*
// check a few parameters
printf("following for pixelint\n");
printf("parameter check samplesPerBurst %d\n",*samplesPerBurst_d);
printf("parameter check range start end %f %f\n",*rngstart_d,*rngend_d);
printf("parameter check dmrg wvl %f %f\n",*dmrg_d,*wvl_d);
printf("parameter check aperture iaperture %d %d\n",*aperture_d,*iaperture_d);
printf("parameter check satloc %f %f %f\n",satloc_d[0],satloc_d[1],satloc_d[2]);
printf("parameter check xyz %f %f %f\n",xyz_d[0],xyz_d[1],xyz_d[2]);
printf("parameter check azoff_d %f %f %f\n",azoff_d[100],azoff_d[1000],azoff_d[1200]);
printf("parameter check burstdata %f %f %f %f\n",hipCrealf(burstdata_d[0]),hipCimagf(burstdata_d[0]),hipCrealf(burstdata_d[1]),hipCimagf(burstdata_d[1]));
printf("above for pixelint\n");
*/

// outer loop is over pixels in line
for (int pixel = index; pixel < *demwidth_d; pixel+=stride){
    //  loop for complex integration
    cacc = make_hipDoubleComplex(0.,0.);

    if (pixeltime_d[pixel] > 0.){
        azstart=round(azoff_d[pixel] - *aperture_d / 2); 
    	azend=round(azoff_d[pixel] + *aperture_d / 2); 
	azstart=max(azstart,0);
	azend=min(azend,*rawdatalines_d-1);
	//printf("azstart azend %d %d\n",azstart,azend);

	for (azline=azstart; azline<azend; azline++){
	  vectx=xyz_d[3 * pixel + 0]-satloc_d[3*azline];
	  vecty=xyz_d[3 * pixel + 1]-satloc_d[3*azline+1];
	  vectz=xyz_d[3 * pixel + 2]-satloc_d[3*azline+2];
	  range=sqrt(vectx*vectx+vecty*vecty+vectz*vectz);
	  if (range>=*rngstart_d && range <= *rngend_d - *dmrg_d){
	    r=(range - *rngstart_d) / *dmrg_d ;
	    intr=floor(r);
	    fracr=r-intr;

	    cval=hipCaddf( hipCmulf(burstdata_d[azline * *samplesPerBurst_d + intr], make_hipComplex(1-fracr,0.)), hipCmulf( burstdata_d[azline * *samplesPerBurst_d + (intr+1)], make_hipFloatComplex(fracr,0.)));

	    phase = 4. * pi / *wvl_d * range;
	    cphase = make_hipDoubleComplex(cos(phase),sin(phase));
	    cacc = hipCadd(cacc, hipCmul(hipComplexFloatToDouble(cval), cphase));
	  }  // end range test 
	}  // end azline loop
    } // end pixel test
    outdata_d[pixel]=hipComplexDoubleToFloat(cacc);

} // end pixel loop

}

extern "C" void azimuth_compress_(float complex *burstdata,double *satloc,int *rawdatalines,int *samplesPerBurst,int *demwidth,int *demlength,int *fdout,int *fddem,double *deltalat,double *deltalon,double *firstlat,double *firstlon, double *latlons,double *timeorbit,double *xx,double *vv,int *numstatevec,double *rngstart,double *rngend,double *tstart,double *tend,double *tmid,double *xyz_mid,double *vel_mid,double *t,double *dtaz,double *dmrg,double *wvl,int *aperture,int *iaperture, double *angc0, double *angc1, double *prf)
{

// internal variables
  double lat;
  double *lon;
  short *demin;
  double *azoff;
  double *pixeltime;
  double tline;
  double rngpix;
  double *xyz;
  double umag, fd, veff, td, udotv;

  int naperture, computeflag; // naperture is integration midpoint in pixels, computeflag sets whether to call integrator
  int pixel,line,i;
  int nbytes;
  long long int iaddr;
  float complex *outdata, *indata;

// variables for openmp pragma argument passing
  int ompdemwidth=*demwidth;
  double omptmid=*tmid;
  double ompangc0=*angc0, ompangc1=*angc1;
  double ompprf=*prf;
  int omprawdatalines=*rawdatalines;
  double omptstart=*tstart, omptend=*tend;
  int ompaperture=*aperture;
  double ompwvl=*wvl;
   
// malloc cpu arrays
  lon = (double *) malloc(*demwidth * sizeof(double));
  demin = (short *) malloc(*demwidth * sizeof(short));
  azoff = (double *) malloc(*demwidth * sizeof(double));
  pixeltime = (double *) malloc(*demwidth * sizeof(double));
  xyz = (double *) malloc(sizeof(double) * 3 * *demwidth);
  outdata = (float complex *)malloc(*demwidth * sizeof(float complex));
  indata = (float complex *)malloc(*demwidth * sizeof(float complex));

// gpu array definitions
   hipFloatComplex *burstdata_d;
   double *satloc_d;

   hipMalloc( (void **)&burstdata_d, sizeof(float complex) * *rawdatalines * *samplesPerBurst);
   hipMalloc( (void **)&satloc_d, sizeof(double) * 3 * *rawdatalines);

// other variables we'll need in the gpu
   double *azoff_d, *xyz_d, *pixeltime_d;
   int    *aperture_d, *iaperture_d, *rawdatalines_d, *samplesPerBurst_d;
   int    *demwidth_d;
   double *rngstart_d, *rngend_d, *dmrg_d, *wvl_d;
   hipFloatComplex *outdata_d;
   double *tstart_d, *tend_d, *tline_d;

   hipMalloc( (void **)&pixeltime_d, sizeof(double) * *demwidth);
   hipMalloc( (void **)&azoff_d, sizeof(double) * *demwidth);
   hipMalloc( (void **)&aperture_d, sizeof(int));
   hipMalloc( (void **)&iaperture_d, sizeof(int));
   hipMalloc( (void **)&rawdatalines_d, sizeof(int));
   hipMalloc( (void **)&xyz_d, sizeof(double) * 3 * *demwidth);
   hipMalloc( (void **)&samplesPerBurst_d, sizeof(int));
   hipMalloc( (void **)&demwidth_d, sizeof(int));
   hipMalloc( (void **)&rngstart_d, sizeof(double));
   hipMalloc( (void **)&rngend_d, sizeof(double));
   hipMalloc( (void **)&dmrg_d, sizeof(double));
   hipMalloc( (void **)&tstart_d, sizeof(double));
   hipMalloc( (void **)&tend_d, sizeof(double));
   hipMalloc( (void **)&tline_d, sizeof(double));
   hipMalloc( (void **)&wvl_d, sizeof(double));
   hipMalloc( (void **)&outdata_d, sizeof(float complex) * *demwidth);

// constants and such
  int LLH_2_XYZ=1;

  double pi,deg2rad;

  struct ellipsoid elp = { 6378137.0, 0.0066943799901499996};
 
  pi = 4. * atan2(1.0,1.0);
  deg2rad = pi/180.0;

//  get some basic gpu architecture info
    int blockSize = 256;
    int numBlocks = (*demwidth+blockSize-1)/blockSize;
    printf("GPU blocksize, numblocks: %d %d\n",blockSize,numBlocks);

//  start setting up shared data arrays in gpu 
  hipMemcpy( burstdata_d, burstdata, sizeof(float complex) * *rawdatalines * *samplesPerBurst, hipMemcpyHostToDevice );
  hipMemcpy( satloc_d, satloc, sizeof(double) * *rawdatalines * 3, hipMemcpyHostToDevice );

//  and other constants needed in pixel integration 
  hipMemcpy( aperture_d, aperture, sizeof(int), hipMemcpyHostToDevice );
  hipMemcpy( iaperture_d, iaperture, sizeof(int), hipMemcpyHostToDevice );
  hipMemcpy( rawdatalines_d, rawdatalines, sizeof(int), hipMemcpyHostToDevice );
  hipMemcpy( samplesPerBurst_d, samplesPerBurst, sizeof(int), hipMemcpyHostToDevice );
  hipMemcpy( demwidth_d, demwidth, sizeof(int), hipMemcpyHostToDevice );
  hipMemcpy( rngstart_d, rngstart, sizeof(double), hipMemcpyHostToDevice );
  hipMemcpy( rngend_d, rngend, sizeof(double), hipMemcpyHostToDevice );
  hipMemcpy( dmrg_d, dmrg, sizeof(double), hipMemcpyHostToDevice );
  hipMemcpy( wvl_d, wvl, sizeof(double), hipMemcpyHostToDevice );

//  begin loop over lines in dem 
  for (i=0;i<*demwidth;i++){
    lon[i]=*firstlon + (i-1)* *deltalon;
  }

  for (line=0;line<*demlength;line++){
    for (i=0;i<*demwidth;i++)outdata[i]=0.+0.*I;
    lat=*firstlat + line * *deltalat;

    if (lat>latlons[0]) {
      if (lat<latlons[1]) {
	//Read in this line from DEM
	iaddr=(line-1)* *demwidth * 2;
	nbytes=lseek(*fddem, (off_t) iaddr, SEEK_SET);
	iaddr= *demwidth * 2;
	nbytes=read(*fddem,demin,iaddr);
	if (nbytes < 0) printf("dem read error %d\n",nbytes);

	if (line % 1000 == 1) printf("Processing line: %d\n",line);
	computeflag=0;

	// parallelize the pixel loop
	#pragma omp parallel for private(pixel,tline,rngpix,umag,udotv,fd,veff,td,naperture) shared(ompdemwidth,lat,deg2rad,demin,elp,LLH_2_XYZ,xyz,omptmid,xyz_mid,vel_mid,timeorbit,xx,vv,numstatevec,ompwvl,pi,ompangc0,ompangc1,ompprf,omprawdatalines,pixeltime,azoff,omptstart,omptend,ompaperture,computeflag)
	for (pixel=0; pixel< ompdemwidth; pixel++){
	// local array definitions for omp loop
	  double *llh = (double *) malloc(sizeof(double)*3);
	  double *xyztemp = (double *) malloc(sizeof(double)*3);
	  double *satx = (double *) malloc(sizeof(double)*3);
	  double *satv = (double *) malloc(sizeof(double)*3);
	  double *unitlookvector = (double *) malloc(sizeof(double)*3);

	  llh[0] = lat * deg2rad;
	  llh[1] = lon[pixel] * deg2rad;
	  llh[2] = demin[pixel];
	  latlon_(&elp,xyztemp,llh,&LLH_2_XYZ);
	  xyz[pixel * 3]=xyztemp[0];
	  xyz[pixel * 3+1]=xyztemp[1];
	  xyz[pixel * 3+2]=xyztemp[2];
	  tline = omptmid;
	  satx[0] = xyz_mid[0];satx[1] = xyz_mid[1];satx[2] = xyz_mid[2];
	  satv[0] = vel_mid[0];satv[1] = vel_mid[1];satv[2] = vel_mid[2];
	  // get the zero doppler location of the satellite
	  orbitrangetime_(xyztemp,timeorbit,xx,vv,numstatevec,&omptmid,satx,satv,&tline,&rngpix);

	  // tops-specific geometry calculations
	  unitlookvector[0]=(xyztemp[0]-satx[0]); unitlookvector[1]=(xyztemp[1]-satx[1]); unitlookvector[2]=(xyztemp[2]-satx[2]);
	  umag=sqrt(unitlookvector[0]*unitlookvector[0]+unitlookvector[1]*unitlookvector[1]+unitlookvector[2]*unitlookvector[2]);
	  unitlookvector[0]=unitlookvector[0]/umag; unitlookvector[1]=unitlookvector[1]/umag; unitlookvector[2]=unitlookvector[2]/umag; 
	  udotv=satv[0]*unitlookvector[0]+satv[1]*unitlookvector[1]+satv[2]*unitlookvector[2];
	  fd=(2.0/ ompwvl) * udotv;
	  veff=sqrt(satv[0]*satv[0]+satv[1]*satv[1]+satv[2]*satv[2]);
	  td=(rngpix* (ompwvl)/2./veff/veff*fd-rngpix*(ompangc0)*pi/180./veff)/(1.+rngpix*(ompangc1)*pi/180./veff);
	  naperture=td*(ompprf) + omprawdatalines/2;
	  
	  pixeltime[pixel]=-999999.;
	  azoff[pixel]=0;
	  if (tline > omptstart-2. && tline < omptend+2.){
	    if (naperture > ompaperture/2 && naperture < omprawdatalines- ompaperture/2){
	      azoff[pixel] = naperture; 
              pixeltime[pixel]=tline;
	      computeflag=1;
	      }
	  }
	  free(llh);
	  free(xyztemp);
	  free(satx);
	  free(satv);
	  free(unitlookvector);
	} // end pixel loop

	if (computeflag == 1){
	// put pixel specific info into gpu
	hipMemcpy( pixeltime_d, pixeltime, sizeof(double) * *demwidth, hipMemcpyHostToDevice );
	hipMemcpy( xyz_d, xyz, sizeof(double) * 3 * *demwidth, hipMemcpyHostToDevice );
	hipMemcpy( azoff_d, azoff, sizeof(double) * *demwidth, hipMemcpyHostToDevice );

	pixelint<<< numBlocks,blockSize >>>(burstdata_d,satloc_d,xyz_d,azoff_d, demwidth_d, aperture_d,iaperture_d,rawdatalines_d,samplesPerBurst_d,rngstart_d,rngend_d,pixeltime_d,dmrg_d,wvl_d,outdata_d);
	hipDeviceSynchronize();
	hipMemcpy( outdata, outdata_d, sizeof(float complex) * *demwidth, hipMemcpyDeviceToHost );

	// get existing data for that line
	iaddr=(line-1)* *demwidth * 8;
	nbytes=lseek(*fdout, (off_t) iaddr, SEEK_SET);
	iaddr= *demwidth * 8;
	nbytes=read(*fdout,indata,iaddr);
	// update if pixel computed
	for (pixel=0; pixel< *demwidth; pixel++){
	    if(abs(creal(outdata[pixel])) > 1.e-18)indata[pixel]=outdata[pixel];
	    }	    
	// write line to file
	iaddr=(line-1)* *demwidth * 8;
	nbytes=lseek(*fdout, (off_t) iaddr, SEEK_SET);
	iaddr= *demwidth * 8;
	nbytes=write(*fdout,indata,iaddr);
	}  // end compute flag test

      } // end if lat in bounds
      
    } // end if lat in bounds #2

  } // end line loop

  // free up memory in gpu
  hipFree(burstdata_d);
  hipFree(satloc_d);
  hipFree(azoff_d);
  hipFree(pixeltime_d);
  hipFree(iaperture_d);
  hipFree(rawdatalines_d);
  hipFree(xyz_d);
  hipFree(samplesPerBurst_d);
  hipFree(rngstart_d);
  hipFree(rngend_d);
  hipFree(dmrg_d);
  hipFree(wvl_d);

}

