//!!!!!!!!!!!!!!
//!
//!  testmultigpu - test using multiple gpus
//!
//!!!!!!!!!!!!!!

#include <stdlib.h>
#include <stdio.h>
#include <complex.h>
#include <math.h>
#include <unistd.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <omp.h>

int main()
{

//  get some basic gpu architecture info
  int blockSize = 256;
  int numBlocks = (1000+blockSize-1)/blockSize;
  printf("GPU blocksize, numblocks: %d %d\n",blockSize,numBlocks);

// get the configuration of the machine
  int nDevices, iDevice;  // multigpu handling parameters
  hipGetDeviceCount(&nDevices);
  printf("Number of gpus: %d\n",nDevices);
  printf("Number of cpus: %d\n",omp_get_num_procs());
  for(iDevice=0;iDevice<nDevices;iDevice++){
    hipDeviceProp_t dprop;
    hipGetDeviceProperties(&dprop,iDevice);
    printf(" %d %s\n",iDevice,dprop.name);
  }

  //  use one cpu thread per device
  omp_set_num_threads(nDevices);

// parallel section
  omp_set_num_threads(nDevices);
#pragma omp parallel
  {
  unsigned int cpu_thread_id = omp_get_thread_num();  // which thread 
  unsigned int num_cpu_threads = omp_get_num_threads();

  // set and check the CUDA device for this CPU thread
  int gpu_id = -1;
  hipSetDevice(cpu_thread_id % nDevices);        // "% nDevices" allows more CPU threads than GPU devices
  hipGetDevice(&gpu_id);
 
  printf("CPU thread %d (of %d) uses CUDA device %d\n", cpu_thread_id, num_cpu_threads, gpu_id);
  
  } // end parallel section

}

