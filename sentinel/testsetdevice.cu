//!!!!!!!!!!!!!!
//!
//!  testsetdevice command
//!
//!!!!!!!!!!!!!!

#include <stdlib.h>
#include <stdio.h>
#include <complex.h>
#include <math.h>
#include <unistd.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <omp.h>

int main()
{

//  get some basic gpu architecture info
  int blockSize = 256;
  int numBlocks = (1000+blockSize-1)/blockSize;
  printf("GPU blocksize, numblocks: %d %d\n",blockSize,numBlocks);

// get the configuration of the machine
  int nDevices, iDevice;  // multigpu handling parameters
  hipGetDeviceCount(&nDevices);
  printf("Number of gpus: %d\n",nDevices);
  for(iDevice=0;iDevice<nDevices;iDevice++){
    hipDeviceProp_t dprop;
    hipGetDeviceProperties(&dprop,iDevice);
    printf(" %d %s\n",iDevice,dprop.name);
  }

  long int ret = hipGetDevice(0);
 
  printf("setdevice returns %d\n", ret);
  

}

